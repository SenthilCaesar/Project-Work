#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>


#define BLOCK_SIZE 32
// include the kernel
#include "matrixMul_kernel.cu"

unsigned Size;
float *A, *B, *C;

void gpuInit(void){
  hipSetDevice(0);
}

int main(int argc, char *argv[]){
  int i, j, k, c;
  float cpuTime, gpuTime;
  hipEvent_t start;
  hipEvent_t stop;

 Size = 1024;
 while ((c = getopt(argc, argv, "s:")) != -1){
    switch (c){
    case 's': sscanf(optarg, "%u", &Size); break;
    default: break;
    }
  }

  size_t memSize = Size * Size * sizeof(float);

  gpuInit();
  hipEventCreate(&start);
  hipEventCreate(&stop);

  A = (float*)malloc(memSize);
  B = (float*)malloc(memSize);
  C = (float*)malloc(memSize);

  printf("matrix size: %d\n", Size);

 for (i=0; i<Size; i++) {
    for (j=0; j<Size; j++){
        if(i <= j) {
                A[i*Size + j] = drand48();
                A[j*Size + i] = A[i*Size + j];
                C[i*Size + j] = 0.0;
          }
   }
}


  hipEventRecord(start, NULL);
#pragma omp parallel for shared(A,B,C,i) private(j,k)
  for (i=0; i<Size; i++){
    for (j=0; j<Size; j++){
      for (k=0; k<Size; k++) {
        if(i <= j) {
        C[i*Size + j] += A[i*Size + k] * A[j*Size + k];
        }
       }
        C[j*Size + i] = C[i*Size + j];
    }
  }


  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&cpuTime, start, stop);
  printf("cpu time: %.3f ms\n", cpuTime);

  float *d_A, *d_B, *d_C;
  hipMalloc((void**) &d_A, memSize);
  hipMalloc((void**) &d_B, memSize);
  hipMalloc((void**) &d_C, memSize);

  hipMemcpy(d_A, A, memSize, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, memSize, hipMemcpyHostToDevice);
  dim3 threads(BLOCK_SIZE, BLOCK_SIZE);                 // 32*32 ( 1024 threads in each block )
  dim3 grid(Size/threads.x, Size/threads.y);            // 4096/32 = 128*128 Blocks = 16,384 blocks and
                                                        // 16,384 * 1024 = 16,777,216
 // There are 32 warps in each block . There are 32 threads in a warp each having consecutive threadIdx
  hipEventRecord(start, NULL);
  matrixMul<<<grid, threads>>>(d_C, d_A, d_B, Size, Size);
 // The follwing would launch 16,384 blocks of 1024 threads each ( total of 16,777,216 threads )
 // threads.x = 32
 // threads.y = 32


hipDeviceSynchronize();
  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);
  hipMemcpy(C, d_C, memSize, hipMemcpyDeviceToHost);
  hipEventElapsedTime(&gpuTime, start, stop);
  printf("gpu time: %.3f ms\n", gpuTime);

//for(int i=0; i < 16; i++) {
//      printf("C[%d] = %f\n", i, C[i]);

//}

  return 0;
}

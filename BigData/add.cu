#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>



// Create two integers for the host
// Allocate memory for copies of them on the device GPU
// Copy the integers to the device memory
// Call the kernel to add them together  <<<   >>>
// Copy the result back to the host memory
// Print out the result that GPU computed
// Free the device memory we allocated


__global__ void Addition(int* a, int* b, int* c) {

        *c = *a + *b;

}

int main()
{
  int a,b,c;
  int *dev_a,*dev_b,*dev_c; // Device Pointers
  int size = sizeof(int);

  hipMalloc((void**)&dev_a, size);
  hipMalloc((void**)&dev_b, size);
  hipMalloc((void**)&dev_c, size);

  a=5,b=6;

  hipMemcpy(dev_a, &a,sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, &b,sizeof(int), hipMemcpyHostToDevice);

  Addition<<< 1,1 >>>(dev_a,dev_b,dev_c);
  hipMemcpy(&c, dev_c,size, hipMemcpyDeviceToHost);

   hipFree(&dev_a);
   hipFree(&dev_b);
   hipFree(&dev_c);

   printf("%d\n", c);
   return 0;

}
